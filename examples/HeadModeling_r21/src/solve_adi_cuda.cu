#include "hip/hip_runtime.h"
#include "grid_point.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>

#include <stdlib.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>
#include <sys/time.h>

#define MAX_DIM 300
#define MAX_CONDS 20

__device__ __constant__ float dcTissuesConds[MAX_CONDS];

typedef struct {
  
  GridPoint *grid;
  int        Nx1, Ny1, Nz1;
  int        Nx, Ny, Nz, N;
  float      Hx, Hy, Hz;
  int        Kmax;
  float      time_step, tol;
  
  int        device;
  float      *PP, *Px, *Py, *Pz;
  int        *sig, *sigy;
  float      *Rx, *Ry, *Rz, *F0, *PP_diff;
  float      *hostPP;
  int        *sigmap, *sigmapy;
  float      varying_tau;
  float      *tissueConds;

  float H1;
  float H2;
  float H3;

} SolverParameters;

inline void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(1);
  }
}

//        x axis

/*
  Each time step is split into three substeps. In each substep we solve a large 
  number of tridiagonal system of equations using  using Thomas algorithm 
  ( http://en.wikipedia.org/wiki/Tridiagonal_matrix_algorithm)

  1) First substep (x-direction computation): solve Ny*Nz independent tri system 
  2) Second substep (y-direction computation): solve Nx*Nz tridiagonal system 
  3) Third substep (z-direction): solve Ny*Nz tridiagonal system

  The amount of computation in the three directions are in the same order, however 
  the memory access pattern is different. 

  The computation must proceed in the above order and all updates are inplace in 
  global memory 

 */

// Here we seperated the computation of F0 = tau * PP[idx] + Rx[idx] + Rz[idx] so 
// that threads memory access is coalesced. and then we write them in a way that 
// SolveX1 will read them coalesced as well    
__global__ void SetF0X(float *F0, float *PP, float *Ry, float *Rz, float tau, 
		       int Nx, int Nyz, int src1, int src2, float srcv1, float srcv2)
{
  __shared__ float shared[16][17]; // avoid bank conflicts
    
  int x0 = blockIdx.x * blockDim.x;
  int y0 = blockIdx.y * blockDim.y;
  int idx1 = Nx  * (y0 + threadIdx.y) + x0 + threadIdx.x;
  int idx2 = Nyz * (x0 + threadIdx.y) + y0 + threadIdx.x;
    
  //  int sidx = (int) (idx1 == src1) + 2 * (int) (idx1 == src2);
  float F = tau * PP[idx1] + (Ry[idx1] + Rz[idx1]);

  if (idx1 == src1) F += srcv1;
  else if (idx1 == src2) F += srcv2;
  shared[threadIdx.y][threadIdx.x] = F;

  syncthreads();
  F0[idx2] = shared[threadIdx.x][threadIdx.y];

}


// Note that SolveX1 reads the conductivity data from a different that is writen 
// in column oreder so that memory access is coalesced

__global__ void SolveX1(float *F0, float *Px, int *sigmap, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, 
			float H1, float tau)
{
  int y    = blockIdx.x * blockDim.x + threadIdx.x;
  int z    = blockIdx.y * blockDim.y + threadIdx.y;    
  int idx0 = y * Nx + z*Ny*Nx;
  int idx2, idx20;
  int idx, i;
  Nx1--;
  
  // These two large (O(256)) local arrays  
  float Al[MAX_DIM];
  float Be[MAX_DIM];

  idx20 = z*Ny*Nx + y;
  float sigma0;
  float sigma1 = dcTissuesConds[sigmap[idx20]];
  float sigma2 = dcTissuesConds[sigmap[idx20+Ny]];
  
  float A, B = H1*(sigma2 + sigma1);
  float al=0, be=0;
  float Zn;
  
  int yz = z*Ny + y;
  int Nyz = Ny*Nz;
    
  // forward calculation 
  for (i=1; i<Nx1; i++) {
    Al[i] = al;
    Be[i] = be;
    
    idx = idx0 + i;
    idx2 = idx20 + i* Ny;

    sigma0 = sigma1;
    sigma1 = sigma2;
    sigma2 = dcTissuesConds[sigmap[idx2+Ny]];

    A = H1*(sigma0 + sigma1);
    B = H1*(sigma2 + sigma1);
        
    Zn = 1.0 / (A + B + tau - al*A);
    be = (A*be + F0[i*Nyz + yz]) * Zn;
    al = B*Zn;
  }
    
  Al[i] = al;
  Be[i] = be;

  Px[idx0 + Nx1] = 0.0;
  float px = 0.0;

  // backward calculation 
  for (i=Nx1-1; i>=0; i--) {
    idx = idx0 + i;
    px = Al[i+1] * px + Be[i+1];
    Px[idx] = px;
  }
}

__global__ void SolveX2(float *Px, float *Rx, int *sigma, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, 
			float H1)
{
  __shared__ float shared_Px[MAX_DIM];
  __shared__ float shared_sigma[MAX_DIM];
    
  int idx   = threadIdx.x;
  int IDX   = blockIdx.y * Nx * Ny + blockIdx.x * Nx + idx;
  //  int sidx  = blockIdx.y * Nx * Ny + blockIdx.x + threadIdx.x * Ny;

  float p1  = Px[IDX];
  float s1  = dcTissuesConds[sigma[IDX]];

  shared_Px[idx] = p1;
  shared_sigma[idx] = s1;
  syncthreads();

    
  if(idx > 0 && idx < Nx1-1) {
    
    float s0 = shared_sigma[idx-1];
    float s2 = shared_sigma[idx+1];
        
    float p0 = shared_Px[idx-1];
    float p2 = shared_Px[idx+1];
    
    float A = H1*(s0 + s1);
    float B = H1*(s2 + s1);
        
    Rx[IDX] = A*p0 - (A+B)*p1 + B*p2;
  }
}

////////////////////////////////////////////////////////////////////////////
/// y axis /////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// Similar to the computation in the x-direction but memory acess is coalesced
__global__ void SolveY1(float *PP, float *Py, float *Rx, float *Rz, int *sigmap, 
			int Nx, int Ny, int Nz, int Nx1, int Ny1, int Nz1, float H2, 
			float tau, int src1, int src2, float srcv1, float srcv2)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
    
  if (x < Nx1 && z < Nz1) {
    Ny1--;
    
    int idx0 = x + z*Ny*Nx;
    int idx, i;
        
    float Al[MAX_DIM];
    float Be[MAX_DIM];
    float al=0, be=0;
    float A, B;
        
    float F0, Zn, sigma0;
    float sigma1 = dcTissuesConds[sigmap[idx0]];
    float sigma2 = dcTissuesConds[sigmap[idx0+Nx]];
        
    for (i=1; i<Ny1; i++) {
      Al[i] = al;
      Be[i] = be;
        
      idx = idx0 + i*Nx;
      
      sigma0 = sigma1;
      sigma1 = sigma2;
      sigma2 = dcTissuesConds[sigmap[idx+Nx]];
            
      A = H2*(sigma0 + sigma1);
      B = H2*(sigma2 + sigma1);

      F0 = tau * PP[idx] + Rx[idx] + Rz[idx];
      if (idx == src1)      F0 += srcv1;
      else if (idx == src2) F0 += srcv2;
      Zn = 1.0 / (A + B + tau - al*A);

      al = B*Zn;
      be = (A*be + F0) * Zn;
    }
        
    Al[i] = al;
    Be[i] = be;
        
    Py[idx0 + Ny1*Nx] = 0.0;
    float py = 0.0;
    
    for (i=Ny1-1; i>=0; i--) {
      idx = idx0 + i*Nx;
      py = Al[i+1] * py + Be[i+1];
      Py[idx] = py;
    }
  }
}

__global__ void SolveY2(float *Ry, float *Py, int *sigma, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, float H2)
{
  int idx  = (blockIdx.y+1) * Nx * Ny + (blockIdx.x + 1) * Nx + threadIdx.x;

  float s0 = dcTissuesConds[sigma[idx-Nx]];
  float s1 = dcTissuesConds[sigma[idx]];
  float s2 = dcTissuesConds[sigma[idx+Nx]];
    
  float A = H2*(s0 + s1);
  float B = H2*(s2 + s1);
    
  float p0 = Py[idx-Nx];
  float p1 = Py[idx];
  float p2 = Py[idx+Nx];
    
  Ry[idx] = A*p0 - (A+B)*p1 + B*p2;
}

////////////////////////////////////////////////////////////////////////////
/// z axis /////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// Similar to the computation in the y-directions with extra computation 
// of the average of potentials computed in the x-, y-, and z-directions
// and the difference of total potential from previous step for convergence 
// check at the end of the time step which require more memory access than 
// the y-direction 

__global__ void SolveZ1(float *PP, float *Px, float *Py, float *Pz, 
			float *Rx, float *Ry, 
			int *sigmap, int Nx, int Ny, int Nz,
			int Nx1, int Ny1, int Nz1, float H3, float tau, 
			float *PP_diff, int src1, int src2, float srcv1, float srcv2)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
    
  if (x < Nx1 && y < Ny1) {
    Nz1--;
    
    int idx0 = x + y*Nx;
    int idx, i;
        
    float Al[MAX_DIM];
    float Be[MAX_DIM];
    float al=0, be=0;
    float A, B;
    
    int Nxy = Nx*Ny;
        
    float F0, Zn, sigma0;
    float sigma1 = dcTissuesConds[sigmap[idx0]];
    float sigma2 = dcTissuesConds[sigmap[idx0+Nxy]];
        
    for (i=1; i<Nz1; i++) {
      Al[i] = al;
      Be[i] = be;
        
      idx = idx0 + i*Nxy;
            
      sigma0 = sigma1;
      sigma1 = sigma2;
      sigma2 = dcTissuesConds[sigmap[idx+Nxy]];
            
      A = H3*(sigma0 + sigma1);
      B = H3*(sigma2 + sigma1);

      //      int sidx = (int) (idx == src1) + 2 * (int) (idx == src2);
      //      F0 = tau * PP[idx] + (Rx[idx] + Ry[idx] + dConstSources[sidx]);

      F0 = tau * PP[idx] + Rx[idx] + Ry[idx];
      if (idx == src1)      F0 += srcv1;
      else if (idx == src2) F0 += srcv2;

      Zn = 1.0 / (A + B + tau - al*A);
            
      al = B*Zn;
      be = (A*be + F0) * Zn;
    }
        
    Al[i] = al;
    Be[i] = be;

    Pz[idx0 + Nz1*Nxy] = 0.0;
    float ppz = 0.0;
    float pp, pz;
        
    for (i=Nz1-1; i>=0; i--) {
      idx = idx0 + i*Nxy;
      
      pz = Al[i+1] * ppz + Be[i+1];
      pp = (Px[idx] + Py[idx] + pz) / 3.0;
      
      // Compute the difference from the previous step 
      // for convergence check at the end of the time step
      PP_diff[idx] = abs(pp - PP[idx]);
      
      PP[idx] = pp;
      Pz[idx] = pz;
      ppz = pz;
    }
  }
}


__global__ void SolveZ2(float *Rz, float *Pz, int *sigma, int Nx, int Ny, int Nz,
			int Nx1, int Ny1, int Nz1, float H3)
{
    int Nxy = Nx*Ny;
    int idx = (blockIdx.y + 1) * Nxy + blockIdx.x * Nx + threadIdx.x;
    
    float s0 = dcTissuesConds[sigma[idx-Nxy]];
    float s1 = dcTissuesConds[sigma[idx]];
    float s2 = dcTissuesConds[sigma[idx+Nxy]];
    
    float A = H3*(s0 + s1);
    float B = H3*(s2 + s1);
    
    float p0 = Pz[idx-Nxy];
    float p1 = Pz[idx];
    float p2 = Pz[idx+Nxy];
    
    Rz[idx] = A*p0 - (A+B)*p1 + B*p2;
}


////////////////////////////////////////////////////////////////////////////
/// main ///////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// reduction function declaration (implemented in reduction.cu)
float full_reduce(float *d_input, float *d_output, unsigned int n, 
		  unsigned int maxThreads, unsigned int maxBlocks);

int InitializeSolver( SolverParameters *arg){

  static int  setdev = 0;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0){
    fprintf(stderr, "There is no device supporting CUDA ");
    return 1;
  }

  char host[200];
  gethostname(host, 200);

  
  if (arg->device < 0 || arg->device >= deviceCount){
    fprintf(stderr, "No such cuda device = %d \n", arg->device);
    return 1;
  }

  else {
    if (!setdev){
      if( hipSetDevice( arg->device ) != hipSuccess) {
	fprintf(stderr, "Failed setting cuda device \n ");
	return 1;
      }
      setdev = 1;
    }
  }

  printf("[%s] using id=%d cuda device.\n", host, arg->device);

  //Make each dimension multiple of 16
  int Nx = arg->Nx1 + 15 - (arg->Nx1-1)%16;
  int Ny = arg->Ny1 + 15 - (arg->Ny1-1)%16;
  int Nz = arg->Nz1 + 15 - (arg->Nz1-1)%16;
  int N  = Nx * Ny * Nz;
    
  // read conductivity values into two flat arrays
  // one writen in row order and the other in column
  int i, j, k, idx0, idx1;
   
  // This is to hold the Potential (the solution)
  arg->hostPP = (float *)malloc(N * sizeof(float));
  
   // row order (x-direction) conductivity array 
  arg->sigmap = (int *)malloc(N * sizeof(int));

  // column order (y-direction) conductivity array 
  arg->sigmapy = (int *)malloc(N * sizeof(int));

  assert(arg->hostPP != NULL  && arg->sigmap != NULL && arg->sigmapy != NULL);
    
  memset(arg->sigmap, 0, N * sizeof(float));
  memset(arg->sigmapy, 0, N * sizeof(float));
  
  int idx2;
  for(k=0; k<arg->Nz1; k++) {
    for(j=0; j<arg->Ny1; j++) {
      for(i=0; i<arg->Nx1; i++) {
	idx0 = k*arg->Nx1*arg->Ny1 + j*arg->Nx1 + i;
	idx1 = k*Nx*Ny + j*Nx + i;
	idx2 = k*Nx*Ny + i*Ny + j;
    
	arg->sigmap[idx1]  = arg->grid[idx0].sigmap - arg->tissueConds;
	arg->sigmapy[idx2] = arg->grid[idx0].sigmap - arg->tissueConds;
      }
    }
  }
  
  arg->Nx = Nx;
  arg->Ny = Ny;
  arg->Nz = Nz;
  arg->N = N;

  // set parameters
  float tau = (arg->Hx+arg->Hy+arg->Hz)/12.0;
  float varying_tau = arg->time_step*(1.0/tau);
  arg->varying_tau = varying_tau;

  float H1 = 0.5/(arg->Hx*arg->Hx);
  float H2 = 0.5/(arg->Hy*arg->Hy);
  float H3 = 0.5/(arg->Hz*arg->Hz);

  arg->H1 = H1;
  arg->H2 = H2;
  arg->H3 = H3;

  // allocate device memory     
  assert(hipMalloc( (void**) &arg->PP,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Px,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Py,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Pz,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Rx,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Ry,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Rz,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->sig,     N*sizeof(int))   == hipSuccess);
  assert(hipMalloc( (void**) &arg->sigy,    N*sizeof(int))   == hipSuccess);
  assert(hipMalloc( (void**) &arg->F0,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->PP_diff, N*sizeof(float)) == hipSuccess);
  checkCUDAError("memory allocation");
    
  hipMemcpy(arg->sig, arg->sigmap, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(arg->sigy, arg->sigmapy, N*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("initialization");
  return 0; 
}

void free_memory(SolverParameters *sp){

  hipFree(sp->PP);
  hipFree(sp->Px);
  hipFree(sp->Py);
  hipFree(sp->Pz);
  hipFree(sp->Rx);
  hipFree(sp->Ry);
  hipFree(sp->Rz);
  hipFree(sp->sig);
  hipFree(sp->sigy);
  hipFree(sp->F0);
  hipFree(sp->PP_diff);

  free(sp->hostPP);
  free(sp->sigmap);
  free(sp->sigmapy);
  
}
  
int multipl16_array(SolverParameters *sp, int pos){
  
  int z = pos/(sp->Nx1*sp->Ny1);
  int rem = pos%(sp->Nx1*sp->Ny1);
  int y = rem/sp->Nx1;
  int x = rem%sp->Nx1;

  return (z*sp->Nx*sp->Ny + y*sp->Nx + x);
}



extern "C" 
int solve_poisson_equation_cuda(GridPoint *grid, int Nx1, int Ny1, int Nz1, 
				float Hx, float Hy, float Hz, 
				float time_step, float tol, int Kmax, int device, 
				float *tissueConds, int num_tissues, int *srcPos, 
				float *srcVal, int done)
{

  static SolverParameters *sp = NULL;

  if (done){
    if (sp != NULL){
      free_memory(sp);
      free(sp);
      sp = NULL;
    }
    return 0;
  }

  if (sp == NULL){
    sp = (SolverParameters*) malloc(sizeof(SolverParameters));
    sp->grid = grid;
    sp->Nx1 = Nx1;
    sp->Ny1 = Ny1;
    sp->Nz1 = Nz1;
    sp->Hx  = Hx;
    sp->Hy  = Hy;
    sp->Hz  = Hz;
    sp->time_step = time_step;
    sp->tol = tol;
    sp->Kmax = Kmax;
    sp->device = device;
    sp->tissueConds = tissueConds;
    InitializeSolver(sp);
  }

  if (num_tissues > MAX_CONDS) {
    fprintf(stderr, "Error copying tissues conds to GPU ... number of tissues exceeds MAX_CONDS \n");
    return 0;
  }

  //  double t0 = getWallTime();
  hipMemcpyToSymbol(HIP_SYMBOL("dcTissuesConds"), tissueConds, num_tissues*sizeof(float));

  // set block sizes
  dim3 dimBlock_F0x(16, 16);
  dim3 dimGrid_F0x(sp->Nx/dimBlock_F0x.x, sp->Ny*sp->Nz/dimBlock_F0x.y);

  dim3 dimBlock_x1(16, 16);
  dim3 dimGrid_x1(sp->Ny/dimBlock_x1.x, sp->Nz/dimBlock_x1.y);
    
  dim3 dimBlock_x2(sp->Nx1);
  dim3 dimGrid_x2(sp->Ny1, sp->Nz1);
    
  dim3 dimBlock_y1(16, 16);
  dim3 dimGrid_y1(sp->Nx/dimBlock_y1.x, sp->Nz/dimBlock_y1.y);
    
  dim3 dimBlock_y2(sp->Nx1);
  dim3 dimGrid_y2(Ny1-2, Nz1-2);

  dim3 dimBlock_z1(16, 16);
  dim3 dimGrid_z1(sp->Nx/dimBlock_z1.x, sp->Ny/dimBlock_z1.y);
    
  dim3 dimBlock_z2(sp->Nx1);
  dim3 dimGrid_z2(sp->Ny1, sp->Nz1-2);

 // copy data to device memory 
  hipMemset(sp->PP, 0, sp->N*sizeof(float));
  hipMemset(sp->Px, 0, sp->N*sizeof(float));
  hipMemset(sp->Py, 0, sp->N*sizeof(float));
  hipMemset(sp->Pz, 0, sp->N*sizeof(float));
  hipMemset(sp->Rx, 0, sp->N*sizeof(float));
  hipMemset(sp->Ry, 0, sp->N*sizeof(float));
  hipMemset(sp->Rz, 0, sp->N*sizeof(float));
  hipMemset(sp->PP_diff, 0, sp->N*sizeof(float));
  checkCUDAError("initialization");
        
  int   iter = 1;
  float pp_diff = FLT_MAX;

  int   src1  = multipl16_array(sp, srcPos[0]);
  int   src2  = multipl16_array(sp, srcPos[1]);
  float srcv1 = srcVal[0];
  float srcv2 = srcVal[1];

  do {

    SetF0X <<< dimGrid_F0x, dimBlock_F0x >>> (sp->F0, sp->PP, sp->Ry, sp->Rz, sp->varying_tau, 
					      sp->Nx, sp->Ny*sp->Nz, src1, src2, srcv1, srcv2);
    checkCUDAError("SetF0X");

    SolveX1 <<< dimGrid_x1, dimBlock_x1 >>> (sp->F0, sp->Px, sp->sigy, sp->Nx, sp->Ny, sp->Nz, 
					     sp->Nx1, sp->Ny1, sp->Nz1, sp->H1, sp->varying_tau);
    checkCUDAError("SolveX1");

    SolveX2 <<< dimGrid_x2, dimBlock_x2 >>> (sp->Px, sp->Rx, sp->sig, sp->Nx, sp->Ny, sp->Nz, 
					     sp->Nx1, sp->Ny1, sp->Nz1, sp->H1);
    checkCUDAError("SolveX2");

    SolveY1 <<< dimGrid_y1, dimBlock_y1 >>> (sp->PP, sp->Py, sp->Rx, sp->Rz, sp->sig, sp->Nx, 
					     sp->Ny, sp->Nz, sp->Nx1, sp->Ny1, sp->Nz1, sp->H2, 
    					     sp->varying_tau, src1, src2, srcv1, srcv2);
    checkCUDAError("SolveY1");

    SolveY2 <<< dimGrid_y2, dimBlock_y2 >>>  (sp->Ry, sp->Py, sp->sig, sp->Nx, sp->Ny, sp->Nz, 
					      sp->Nx1, sp->Ny1, sp->Nz1, sp->H2);
    checkCUDAError("SolveY2");

    SolveZ1 <<< dimGrid_z1, dimBlock_z1 >>> (sp->PP, sp->Px, sp->Py, sp->Pz, sp->Rx, sp->Ry, 
					     sp->sig, sp->Nx, sp->Ny, sp->Nz, sp->Nx1, sp->Ny1, 
					     sp->Nz1, sp->H3, sp->varying_tau, sp->PP_diff, src1, 
					     src2, srcv1, srcv2);
    checkCUDAError("SolveZ1");

    SolveZ2 <<< dimGrid_z2, dimBlock_z2 >>> (sp->Rz, sp->Pz, sp->sig, sp->Nx, sp->Ny, sp->Nz, sp->Nx1, 
					     sp->Ny1, sp->Nz1, sp->H3);
    checkCUDAError("SolveZ2");
        
    pp_diff = full_reduce(sp->PP_diff, sp->PP_diff, sp->N, 512, 256);
    checkCUDAError("reduce");

    hipMemset(sp->PP_diff, 0, sp->N*sizeof(float));


  } while(++iter <= sp->Kmax && pp_diff > sp->tol);

  //  printf("termination: %d %d %f %f \n", iter, sp->Kmax, pp_diff, sp->tol);
    
  // copy data back to host and free device memory
  // assert(hipMemcpy(sp->hostPP, sp->PP, sp->N*sizeof(float), hipMemcpyDeviceToHost)  == hipSuccess);

  hipMemcpy(sp->hostPP, sp->PP, sp->N*sizeof(float), hipMemcpyDeviceToHost);
  checkCUDAError("copy");

  //  if (iter < 3) 
  //    fprintf(stderr, "Error computing on device %d \n", device);


  int k, j, i, idx0, idx1;

  // copy solution from flat array back into array of structs
  for(k=0; k<sp->Nz1; k++) {
    for(j=0; j<sp->Ny1; j++) {
      for(i=0; i<sp->Nx1; i++) {
	idx0 = k*sp->Nx1*sp->Ny1 + j*sp->Nx1 + i;
	idx1 = k*sp->Nx*sp->Ny + j*sp->Nx + i;
	sp->grid[idx0].PP = sp->hostPP[idx1];
	if (isnan(sp->grid[idx0].PP)){
	  fprintf(stderr, "Solve_adi_cuda: Invalid solution\n");
	  exit(1);
	}
      }
    }
  }

  //   printf("Iterative Loop time = %d %f\n ", iter, getWallTime()-t0);

  checkCUDAError("finalization");
  return iter;
}
