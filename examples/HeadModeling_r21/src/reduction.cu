#include "hip/hip_runtime.h"
#include <stdio.h>
#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif


extern "C" int get_cuda_device_count(){

  int dev = 0;
  int deviceCount;

  hipError_t cudaResultCode;

  hipGetDeviceCount(&deviceCount);
  cudaResultCode =  hipGetLastError();

  if (cudaResultCode != hipSuccess || deviceCount == 0)
    return 0;

  else{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    if (deviceProp.major == 9999 && deviceProp.minor == 9999)
      return 0;
    else
      return deviceCount;
  }
}


__global__ void vecAdd(int *A,int *B,int *C,int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
}

extern "C" bool test_cuda_device(int deviceId){

  if( hipSetDevice( deviceId ) != hipSuccess) {
    fprintf(stderr, "Failed setting cuda device \n ");
    return false;
  }

  int N = 10;
  int *ah = new int[N]; 
  int *bh = new int[N];
  int *ch = new int[N];

  int *ad,*bd,*cd;

  int block_size = N;
  int num_blocks = N/block_size;

  dim3 dimBlock(block_size, 1, 1);
  dim3 dimGrid(num_blocks, 1, 1);

  for(int i=0; i<N; i++) ah[i] = bh[i] = i;

  //Allocating memory on device
  hipMalloc((void **)&ad, N*sizeof(int));
  hipMalloc((void **)&bd, N*sizeof(int));
  hipMalloc((void **)&cd, N*sizeof(int));

  //copying the arrays from host to device
  hipMemcpy(ad, ah, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(bd, bh, N*sizeof(int), hipMemcpyHostToDevice);

  //add the two vector on device
  vecAdd<<<num_blocks, block_size>>>(ad,bd,cd,N);
  hipDeviceSynchronize();

  //copying the sum back from device to host
  hipMemcpy(ch, cd, N*sizeof(int), hipMemcpyDeviceToHost);

  bool ok = true;
  for (int i=0; i<N; i++){
    // printf("%d :: %d = %d\n", i,  ch[i], ah[i]+bh[i]);
    if (ch[i] != ah[i]+bh[i]) {
      ok = false;
      break;
    }
  }

  hipFree(ad);
  hipFree(bd);
  hipFree(cd);
  return ok;
}


unsigned int nextPow2(unsigned int x) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

template <unsigned int blockSize>
__global__ void reduce_kernel(float *d_input, float *d_output, unsigned int n) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;
  sdata[tid] = 0;

  while (i < n) {

      sdata[tid] = fmaxf(sdata[tid], d_input[i]);
      if (i + blockSize < n) 
	sdata[tid] = fmaxf(sdata[tid], d_input[i+blockSize]);
            
      i += gridSize;
    } 
  __syncthreads();

  if (blockSize >= 1024) { if (tid < 512) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 512]); } __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { sdata[tid] = fmaxf(sdata[tid], sdata[tid +  64]); } __syncthreads(); }
  if (blockSize >= 1024){ if (tid < 512) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 512]); } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
  if (tid < 32)
#endif
    {
      volatile float* smem = sdata;
      if (blockSize >=  64) { smem[tid] = fmaxf(smem[tid], smem[tid + 32]); EMUSYNC; }
      if (blockSize >=  32) { smem[tid] = fmaxf(smem[tid], smem[tid + 16]); EMUSYNC; }
      if (blockSize >=  16) { smem[tid] = fmaxf(smem[tid], smem[tid +  8]); EMUSYNC; }
      if (blockSize >=   8) { smem[tid] = fmaxf(smem[tid], smem[tid +  4]); EMUSYNC; }
      if (blockSize >=   4) { smem[tid] = fmaxf(smem[tid], smem[tid +  2]); EMUSYNC; }
      if (blockSize >=   2) { smem[tid] = fmaxf(smem[tid], smem[tid +  1]); EMUSYNC; }
    }
  if (tid == 0)
    d_output[blockIdx.x] = sdata[0];
}

void partial_reduce(float *d_input, float *d_output, unsigned int n, unsigned int threads, unsigned int blocks)
{
  switch(threads) {
  case 1024:   reduce_kernel<1024> <<<blocks, 1024, 1024*sizeof(float)>>> (d_input, d_output, n);   break;
  case 512:   reduce_kernel<512> <<<blocks, 512, 512*sizeof(float)>>> (d_input, d_output, n);   break;
  case 256:   reduce_kernel<256> <<<blocks, 256, 256*sizeof(float)>>> (d_input, d_output, n);   break;
  case 128:   reduce_kernel<128> <<<blocks, 128, 128*sizeof(float)>>> (d_input, d_output, n);   break;
  case 64:    reduce_kernel<64>  <<<blocks,  64,  64*sizeof(float)>>> (d_input, d_output, n);   break;
  case 32:    reduce_kernel<32>  <<<blocks,  32,  32*sizeof(float)>>> (d_input, d_output, n);   break;
  case 16:    reduce_kernel<16>  <<<blocks,  16,  16*sizeof(float)>>> (d_input, d_output, n);   break;
  case 8:     reduce_kernel<8>   <<<blocks,   8,   8*sizeof(float)>>> (d_input, d_output, n);   break;
  case 4:     reduce_kernel<4>   <<<blocks,   4,   4*sizeof(float)>>> (d_input, d_output, n);   break;
  case 2:     reduce_kernel<2>   <<<blocks,   2,   2*sizeof(float)>>> (d_input, d_output, n);   break;
  case 1:     reduce_kernel<1>   <<<blocks,   1,   1*sizeof(float)>>> (d_input, d_output, n);   break;
  }
}

float full_reduce(float *d_input, float *d_output, unsigned int n, unsigned int maxThreads, unsigned int maxBlocks)
{

  unsigned int threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
  unsigned int blocks = min((n + (threads * 2 - 1)) / (threads * 2), maxBlocks);
  partial_reduce(d_input, d_output, n, threads, blocks);
  n = blocks;


  while(n > 1) {
    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks = min((n + (threads * 2 - 1)) / (threads * 2), maxBlocks);

    partial_reduce(d_output, d_output, n, threads, blocks);
    n = blocks;
  }
    
  float result;
  hipMemcpy(&result, d_output, sizeof(float), hipMemcpyDeviceToHost);
  return result;

}
