#include "hip/hip_runtime.h"
/*

Cuda implementation of the VAI forward solver 

Optimization 2-b: Here we switched to float4 and int4 data types 
                  for more effecient memory access and computing indices.
		  Also, loop unrolling in computing the neighboring elements 
		  contribution is dones is done

Optimization 3  : Since we only use two or few current sources, here I 
                  replaced the sources array by passing the sources as parameters 
		  or copy them to constant memory. Whether there is a current source at
		  an element or not is encoded as the first digit in the first neigbing 
		  idices list
 
		  The redundant elements array is removed 
		  GPU memory usage is about 700MB

Adnan Salman: 11/1/2011

*/

#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <unistd.h>


#define THREADS_PER_BLOCK 32
#define HOST_NAME_SIZE    200

// reduction function declaration (implemented in reduction.cu)
float full_reduce(float *d_input, float *d_output, unsigned int n, 
		  unsigned int maxThreads, unsigned int maxBlocks);


__device__ float DotProdVec4(float4 A, float4 B){
  return (A.x*B.x + A.y*B.y + A.z*B.z + A.w*B.w);
}

__device__ float ComputeFF(int idx, int neighborElemIdx, float4 *d4_v1, int inElementsSize, 
			   float dvv1, float4 *d4_AFF, float Tss, float du3, int j){

    float ss = 0,  uvv;

    if (neighborElemIdx != inElementsSize) {

      int    idx0  = idx*16+2*j;
      float4 ndv11 = d4_v1[neighborElemIdx*2];
      float4 ndv12 = d4_v1[neighborElemIdx*2 + 1];
      float ndv[8] = {ndv11.x, ndv11.y, ndv11.z, ndv11.w, 
		      ndv12.x, ndv12.y, ndv12.z, ndv12.w};
    
      uvv = (dvv1 + ndv[7-j])/2.0;
      ss = DotProdVec4(d4_AFF[idx0], ndv11) + DotProdVec4(d4_AFF[idx0+1], ndv12);

    }
    else uvv = dvv1;
    
    return (Tss * uvv +  du3 - ss);
}

__global__ void  DevComputeff4(int inElementsSize, int4 *d4_IJNZ, float4 *d4_srcs,
			       float4 *d4_AFF, float4 *d4_v1, float4 *ff4, float Tss){

  int idx  = blockIdx.x * blockDim.x + threadIdx.x;  // element index
  int idx0 = idx*2;                                  // type4 data index 

  if (idx <inElementsSize){
    int4 dijnz1 = d4_IJNZ[idx0];                    
    float4 dv11 = d4_v1[idx0];
    float4 du31 = make_float4(0,0,0,0);
    float4 du32 = make_float4(0,0,0,0);

    int srcid = dijnz1.x % 10;
    if (srcid){
      du31 = d4_srcs[(srcid-1)*2];
      du32 = d4_srcs[(srcid-1)*2+1];
    }

    float4 ff1;

    //make sure all argument are the same
    ff1.x = ComputeFF(idx, dijnz1.x/10, d4_v1, inElementsSize, dv11.x, d4_AFF, Tss, du31.x, 0);
    ff1.y = ComputeFF(idx, dijnz1.y, d4_v1, inElementsSize, dv11.y, d4_AFF, Tss, du31.y, 1);
    ff1.z = ComputeFF(idx, dijnz1.z, d4_v1, inElementsSize, dv11.z, d4_AFF, Tss, du31.z, 2);
    ff1.w = ComputeFF(idx, dijnz1.w, d4_v1, inElementsSize, dv11.w, d4_AFF, Tss, du31.w, 3);
    ff4[idx0] = ff1;

    dijnz1 = d4_IJNZ[idx0+1];
    dv11   = d4_v1[idx0+1];
    ff1.x = ComputeFF(idx, dijnz1.x, d4_v1, inElementsSize, dv11.x, d4_AFF, Tss, du32.x, 4);
    ff1.y = ComputeFF(idx, dijnz1.y, d4_v1, inElementsSize, dv11.y, d4_AFF, Tss, du32.y, 5);
    ff1.z = ComputeFF(idx, dijnz1.z, d4_v1, inElementsSize, dv11.z, d4_AFF, Tss, du32.z, 6);
    ff1.w = ComputeFF(idx, dijnz1.w, d4_v1, inElementsSize, dv11.w, d4_AFF, Tss, du32.w, 7);
    ff4[idx0+1] = ff1;

  }
}

__global__ void  UpdateSolution(int inElementsSize, float4 *d4_AL, float4 *d4_v1, float4 *ff4, float4 *diff4_v1){

  //TODO: use the temproray ff4 array to hold the temprary diff4_v1 data 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float4 dff1 = ff4[idx*2];
  float4 dff2 = ff4[idx*2+1];
  float4 dvvv;
  int idx0 = idx*16;

  float4 diffv1 = d4_v1[idx*2];

  dvvv.x = DotProdVec4(d4_AL[idx0],   dff1) + DotProdVec4(d4_AL[idx0+1], dff2);
  dvvv.y = DotProdVec4(d4_AL[idx0+2], dff1) + DotProdVec4(d4_AL[idx0+2+1], dff2);
  dvvv.z = DotProdVec4(d4_AL[idx0+4], dff1) + DotProdVec4(d4_AL[idx0+4+1], dff2);
  dvvv.w = DotProdVec4(d4_AL[idx0+6], dff1) + DotProdVec4(d4_AL[idx0+6+1], dff2);
  d4_v1[idx*2] = dvvv;

  diffv1.x = fabs(diffv1.x - dvvv.x);
  diffv1.y = fabs(diffv1.y - dvvv.y);
  diffv1.z = fabs(diffv1.z - dvvv.z);
  diffv1.w = fabs(diffv1.w - dvvv.w);
  diff4_v1[idx*2] = diffv1;

  dvvv.x = DotProdVec4(d4_AL[idx0+8],  dff1) + DotProdVec4(d4_AL[idx0+8+1], dff2);
  dvvv.y = DotProdVec4(d4_AL[idx0+10], dff1) + DotProdVec4(d4_AL[idx0+10+1], dff2);
  dvvv.z = DotProdVec4(d4_AL[idx0+12], dff1) + DotProdVec4(d4_AL[idx0+12+1], dff2);
  dvvv.w = DotProdVec4(d4_AL[idx0+14], dff1) + DotProdVec4(d4_AL[idx0+14+1], dff2);

  diffv1 = d4_v1[idx*2+1];
  d4_v1[idx*2+1] = dvvv;

  diffv1.x = fabs(diffv1.x - dvvv.x);
  diffv1.y = fabs(diffv1.y - dvvv.y);
  diffv1.z = fabs(diffv1.z - dvvv.z);
  diffv1.w = fabs(diffv1.w - dvvv.w);
  diff4_v1[idx*2+1] = diffv1;

}

int InitializeSolver( int device){

  //TODO: move all cuda initialization here 
  
  static int setdev = 0;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0){
    fprintf(stderr, "There is no device supporting CUDA ");
    return 1;
  }
  
  if (device < 0 || device >= deviceCount){
    fprintf(stderr, "No such cuda device = %d \n", device);
    return 1;
  }

  else {
    if (!setdev){
      if( hipSetDevice( device ) != hipSuccess) {
	fprintf(stderr, "Failed setting cuda device \n ");
	return 1;
      }
      setdev = 1;
    }
  }

  char host[HOST_NAME_SIZE];
  gethostname(host, HOST_NAME_SIZE);

  printf("[%s] using id=%d cuda device.\n", host, device);

  return 0; 

}

extern "C" int SolveVaiCuda(int inElementsSize, int* IJNZ, float *v1, float *AFF, float *AL,
			    int device, float Tss, float *srcs, int num_srcs, float scaledTol, 
			    int maxNumIterations, int checkMax, float eps, int printFlag){

  static int setdev = 0;
  if (setdev == 0){
    InitializeSolver(device);
    setdev = 1;
  }

  int   iterations, check = 0; 
  float loop_step_diff = FLT_MAX, diff = FLT_MAX;

  dim3 blocks(THREADS_PER_BLOCK);
  dim3 grid(inElementsSize/blocks.x + 1);

  float4 *d4_v1, *d4_AL, *d4_AFF, *d4_ff, *d4_srcs, *diff4_v1;
  int4   *d4_IJNZ;

  float * ff = (float *) malloc(inElementsSize*8*sizeof(float));

  assert( hipMalloc( (void**) &d4_IJNZ, inElementsSize*2*sizeof(int4))   == hipSuccess);
  assert( hipMemcpy( d4_IJNZ, IJNZ, inElementsSize*8*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);

  assert( hipMalloc( (void**) &d4_AFF, inElementsSize*16*sizeof(float4))   == hipSuccess);
  assert( hipMemcpy( d4_AFF, AFF, inElementsSize*64*sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

  assert( hipMalloc( (void**) &d4_AL, inElementsSize*16*sizeof(float4))   == hipSuccess);
  assert( hipMemcpy( d4_AL, AL, inElementsSize*64*sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

  assert( hipMalloc( (void**) &d4_ff, inElementsSize*2*sizeof(float4))   == hipSuccess);

  assert( hipMalloc( (void**) &d4_srcs, num_srcs*2*sizeof(float4))   == hipSuccess);
  assert( hipMemcpy( d4_srcs, srcs, num_srcs*8*sizeof(float), hipMemcpyHostToDevice) == hipSuccess);

  assert( hipMalloc( (void**) &d4_v1, inElementsSize*2*sizeof(float4))   == hipSuccess);
  assert( hipMemset( d4_v1, 0, inElementsSize*2*sizeof(float4)) == hipSuccess);

  assert( hipMalloc( (void**) &diff4_v1, inElementsSize*2*sizeof(float4))   == hipSuccess);
  assert( hipMemset( diff4_v1, -1, inElementsSize*8*sizeof(float)) == hipSuccess);

  ///temp stuff
  assert( hipMemset( d4_ff, 0,  inElementsSize*8*sizeof(float)) == hipSuccess);
  
  for(iterations = 0; iterations < maxNumIterations && loop_step_diff > scaledTol && 
	check < checkMax; iterations++) {

    DevComputeff4<<<grid, blocks>>>(inElementsSize, d4_IJNZ, d4_srcs, d4_AFF, d4_v1, d4_ff, Tss);
    UpdateSolution<<<grid, blocks>>> (inElementsSize, d4_AL, d4_v1, d4_ff, diff4_v1);

    loop_step_diff = full_reduce((float*)diff4_v1, (float*) diff4_v1, inElementsSize*8, 512, 256);
    if (fabs(loop_step_diff - diff) < eps ) check++;
    else {
      check = 0;
      diff = loop_step_diff;
    }
  }

  assert(hipMemcpy(v1, d4_v1, (inElementsSize)*2*sizeof(float4), hipMemcpyDeviceToHost) == hipSuccess);

  /*
  hipError_t err = hipMemcpy(v1, d4_v1, (inElementsSize)*2*sizeof(float4), hipMemcpyDeviceToHost);

  if (err ==     hipSuccess)
    printf("Success ");
  else if (err == hipErrorInvalidValue)
    printf("hipErrorInvalidValue ");
  else if (err == hipErrorInvalidDevicePointer)
    printf("hipErrorInvalidDevicePointer ");
  else if (err == hipErrorInvalidMemcpyDirection)
    printf("hipErrorInvalidMemcpyDirection ");
  else 
    printf("Unknown error ");
  */


  hipDeviceSynchronize();

  hipFree(d4_IJNZ);
  hipFree(d4_AFF);
  hipFree(d4_AL);
  hipFree(d4_ff);
  hipFree(d4_srcs);
  hipFree(diff4_v1);
  hipFree(d4_v1);

  return iterations;
  
}
