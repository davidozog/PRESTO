#include "hip/hip_runtime.h"
#include "grid_point.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>

#include <stdlib.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>
#include <sys/time.h>

#define MAX_DIM 300
#define MAX_CONDS 20

/* 
   This a copy of solve_adi_cuda.cu that supports 
   the use of BMD as well. However 
   when not using BMD the oreginal copy is might be more effecient 
   
   meanwhile, both copies are used, but will merge them in a better way later

*/


typedef struct {
  
  GridPoint  *grid;
  int         Nx1, Ny1, Nz1;
  int         Nx, Ny, Nz, N;
  float       Hx, Hy, Hz;
  int         Kmax;
  float       time_step, tol;
  
  int         device;
  float      *PP, *Px, *Py, *Pz;
  float      *sig, *sigy;
  float      *Rx, *Ry, *Rz, *F0, *PP_diff;
  float      *hostPP;
  float      *sigmap, *sigmapy;
  float       varying_tau;
  float      *tissueConds;

  float       H1;
  float       H2;
  float       H3;

} SolverParametersBmd;


inline void checkCUDAErrorBmd(const char *msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(1);
  }
}

// Here we seperated the computation of F0 = tau * PP[idx] + Rx[idx] + Rz[idx] so 
// that threads memory access is coalesced. and then we write them in a way that 
// SolveX1 will read them coalesced as well    
__global__ void SetF0XBmd(float *F0, float *PP, float *Ry, float *Rz, float tau, 
		       int Nx, int Nyz, int src1, int src2, float srcv1, float srcv2)
{
  __shared__ float shared[16][17]; // avoid bank conflicts
    
  int x0 = blockIdx.x * blockDim.x;
  int y0 = blockIdx.y * blockDim.y;
  int idx1 = Nx  * (y0 + threadIdx.y) + x0 + threadIdx.x;
  int idx2 = Nyz * (x0 + threadIdx.y) + y0 + threadIdx.x;

  float F = tau * PP[idx1] + (Ry[idx1] + Rz[idx1]);

  if (idx1 == src1) F += srcv1;
  else if (idx1 == src2) F += srcv2;
  shared[threadIdx.y][threadIdx.x] = F;

  syncthreads();
  F0[idx2] = shared[threadIdx.x][threadIdx.y];

}


// Note that SolveX1 reads the conductivity data from a different that is writen 
// in column oreder so that memory access is coalesced

__global__ void SolveX1Bmd(float *F0, float *Px, float *sigmap, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, 
			float H1, float tau)
{
  int y    = blockIdx.x * blockDim.x + threadIdx.x;
  int z    = blockIdx.y * blockDim.y + threadIdx.y;    
  int idx0 = y * Nx + z*Ny*Nx;
  int idx2, idx20;
  int idx, i;
  Nx1--;
  
  // These two large (O(256)) local arrays  
  float Al[MAX_DIM];
  float Be[MAX_DIM];

  idx20 = z*Ny*Nx + y;
  float sigma0;
  float sigma1 = sigmap[idx20];
  float sigma2 = sigmap[idx20+Ny];
  
  float A, B = H1*(sigma2 + sigma1);
  float al=0, be=0;
  float Zn;
  
  int yz = z*Ny + y;
  int Nyz = Ny*Nz;
    
  // forward calculation 
  for (i=1; i<Nx1; i++) {
    Al[i] = al;
    Be[i] = be;
    
    idx = idx0 + i;
    idx2 = idx20 + i* Ny;

    sigma0 = sigma1;
    sigma1 = sigma2;
    sigma2 = sigmap[idx2+Ny];

    A = H1*(sigma0 + sigma1);
    B = H1*(sigma2 + sigma1);
        
    Zn = 1.0 / (A + B + tau - al*A);
    be = (A*be + F0[i*Nyz + yz]) * Zn;
    al = B*Zn;
  }
    
  Al[i] = al;
  Be[i] = be;

  Px[idx0 + Nx1] = 0.0;
  float px = 0.0;

  // backward calculation 
  for (i=Nx1-1; i>=0; i--) {
    idx = idx0 + i;
    px = Al[i+1] * px + Be[i+1];
    Px[idx] = px;
  }
}

__global__ void SolveX2Bmd(float *Px, float *Rx, float *sigma, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, 
			float H1)
{
  __shared__ float shared_Px[MAX_DIM];
  __shared__ float shared_sigma[MAX_DIM];
    
  int idx   = threadIdx.x;
  int IDX   = blockIdx.y * Nx * Ny + blockIdx.x * Nx + idx;
  //  int sidx  = blockIdx.y * Nx * Ny + blockIdx.x + threadIdx.x * Ny;

  float p1  = Px[IDX];
  float s1  = sigma[IDX];

  shared_Px[idx] = p1;
  shared_sigma[idx] = s1;
  syncthreads();

    
  if(idx > 0 && idx < Nx1-1) {
    
    float s0 = shared_sigma[idx-1];
    float s2 = shared_sigma[idx+1];
        
    float p0 = shared_Px[idx-1];
    float p2 = shared_Px[idx+1];
    
    float A = H1*(s0 + s1);
    float B = H1*(s2 + s1);
        
    Rx[IDX] = A*p0 - (A+B)*p1 + B*p2;
  }
}

////////////////////////////////////////////////////////////////////////////
/// y axis /////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// Similar to the computation in the x-direction but memory acess is coalesced
__global__ void SolveY1Bmd(float *PP, float *Py, float *Rx, float *Rz, float *sigmap, 
			int Nx, int Ny, int Nz, int Nx1, int Ny1, int Nz1, float H2, 
			float tau, int src1, int src2, float srcv1, float srcv2)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
    
  if (x < Nx1 && z < Nz1) {
    Ny1--;
    
    int idx0 = x + z*Ny*Nx;
    int idx, i;
        
    float Al[MAX_DIM];
    float Be[MAX_DIM];
    float al=0, be=0;
    float A, B;
        
    float F0, Zn, sigma0;
    float sigma1 = sigmap[idx0];
    float sigma2 = sigmap[idx0+Nx];
        
    for (i=1; i<Ny1; i++) {
      Al[i] = al;
      Be[i] = be;
        
      idx = idx0 + i*Nx;
      
      sigma0 = sigma1;
      sigma1 = sigma2;
      sigma2 = sigmap[idx+Nx];
            
      A = H2*(sigma0 + sigma1);
      B = H2*(sigma2 + sigma1);

      F0 = tau * PP[idx] + Rx[idx] + Rz[idx];
      if (idx == src1)      F0 += srcv1;
      else if (idx == src2) F0 += srcv2;
      Zn = 1.0 / (A + B + tau - al*A);

      al = B*Zn;
      be = (A*be + F0) * Zn;
    }
        
    Al[i] = al;
    Be[i] = be;
        
    Py[idx0 + Ny1*Nx] = 0.0;
    float py = 0.0;
    
    for (i=Ny1-1; i>=0; i--) {
      idx = idx0 + i*Nx;
      py = Al[i+1] * py + Be[i+1];
      Py[idx] = py;
    }
  }
}

__global__ void SolveY2Bmd(float *Ry, float *Py, float *sigma, int Nx, 
			int Ny, int Nz, int Nx1, int Ny1, int Nz1, float H2)
{
  int idx  = (blockIdx.y+1) * Nx * Ny + (blockIdx.x + 1) * Nx + threadIdx.x;

  float s0 = sigma[idx-Nx];
  float s1 = sigma[idx];
  float s2 = sigma[idx+Nx];
    
  float A = H2*(s0 + s1);
  float B = H2*(s2 + s1);
    
  float p0 = Py[idx-Nx];
  float p1 = Py[idx];
  float p2 = Py[idx+Nx];
    
  Ry[idx] = A*p0 - (A+B)*p1 + B*p2;
}

////////////////////////////////////////////////////////////////////////////
/// z axis /////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// Similar to the computation in the y-directions with extra computation 
// of the average of potentials computed in the x-, y-, and z-directions
// and the difference of total potential from previous step for convergence 
// check at the end of the time step which require more memory access than 
// the y-direction 

__global__ void SolveZ1Bmd(float *PP, float *Px, float *Py, float *Pz, 
			float *Rx, float *Ry, 
			float *sigmap, int Nx, int Ny, int Nz,
			int Nx1, int Ny1, int Nz1, float H3, float tau, 
			float *PP_diff, int src1, int src2, float srcv1, float srcv2)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
    
  if (x < Nx1 && y < Ny1) {
    Nz1--;
    
    int idx0 = x + y*Nx;
    int idx, i;
        
    float Al[MAX_DIM];
    float Be[MAX_DIM];
    float al=0, be=0;
    float A, B;
    
    int Nxy = Nx*Ny;
        
    float F0, Zn, sigma0;
    float sigma1 = sigmap[idx0];
    float sigma2 = sigmap[idx0+Nxy];
        
    for (i=1; i<Nz1; i++) {
      Al[i] = al;
      Be[i] = be;
        
      idx = idx0 + i*Nxy;
            
      sigma0 = sigma1;
      sigma1 = sigma2;
      sigma2 = sigmap[idx+Nxy];
            
      A = H3*(sigma0 + sigma1);
      B = H3*(sigma2 + sigma1);

      //      int sidx = (int) (idx == src1) + 2 * (int) (idx == src2);
      //      F0 = tau * PP[idx] + (Rx[idx] + Ry[idx] + dConstSources[sidx]);

      F0 = tau * PP[idx] + Rx[idx] + Ry[idx];
      if (idx == src1)      F0 += srcv1;
      else if (idx == src2) F0 += srcv2;

      Zn = 1.0 / (A + B + tau - al*A);
            
      al = B*Zn;
      be = (A*be + F0) * Zn;
    }
        
    Al[i] = al;
    Be[i] = be;

    Pz[idx0 + Nz1*Nxy] = 0.0;
    float ppz = 0.0;
    float pp, pz;
        
    for (i=Nz1-1; i>=0; i--) {
      idx = idx0 + i*Nxy;
      
      pz = Al[i+1] * ppz + Be[i+1];
      pp = (Px[idx] + Py[idx] + pz) / 3.0;
      
      // Compute the difference from the previous step 
      // for convergence check at the end of the time step
      PP_diff[idx] = abs(pp - PP[idx]);
      
      PP[idx] = pp;
      Pz[idx] = pz;
      ppz = pz;
    }
  }
}


__global__ void SolveZ2Bmd(float *Rz, float *Pz, float *sigma, int Nx, int Ny, int Nz,
			int Nx1, int Ny1, int Nz1, float H3)
{
    int Nxy = Nx*Ny;
    int idx = (blockIdx.y + 1) * Nxy + blockIdx.x * Nx + threadIdx.x;
    
    float s0 = sigma[idx-Nxy];
    float s1 = sigma[idx];
    float s2 = sigma[idx+Nxy];
    
    float A = H3*(s0 + s1);
    float B = H3*(s2 + s1);
    
    float p0 = Pz[idx-Nxy];
    float p1 = Pz[idx];
    float p2 = Pz[idx+Nxy];
    
    Rz[idx] = A*p0 - (A+B)*p1 + B*p2;
}


////////////////////////////////////////////////////////////////////////////
/// main ///////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

// reduction function declaration (implemented in reduction.cu)
float full_reduce(float *d_input, float *d_output, unsigned int n, 
		  unsigned int maxThreads, unsigned int maxBlocks);

int InitializeSolverBmd( SolverParametersBmd *arg){

  static int  setdev = 0;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0){
    fprintf(stderr, "There is no device supporting CUDA ");
    return 1;
  }

  char host[200];
  gethostname(host, 200);

  
  if (arg->device < 0 || arg->device >= deviceCount){
    fprintf(stderr, "No such cuda device = %d \n", arg->device);
    return 1;
  }

  else {
    if (!setdev){
      if( hipSetDevice( arg->device ) != hipSuccess) {
	fprintf(stderr, "Failed setting cuda device \n ");
	return 1;
      }
      setdev = 1;
    }
  }

  printf("[%s] using id=%d cuda device.\n", host, arg->device);

  //Make each dimension multiple of 16
  int Nx = arg->Nx1 + 15 - (arg->Nx1-1)%16;
  int Ny = arg->Ny1 + 15 - (arg->Ny1-1)%16;
  int Nz = arg->Nz1 + 15 - (arg->Nz1-1)%16;
  int N  = Nx * Ny * Nz;
    
  // read conductivity values into two flat arrays
  // one writen in row order and the other in column
  int i, j, k, idx0, idx1;
   
  // This is to hold the Potential (the solution)
  arg->hostPP = (float *)malloc(N * sizeof(float));
  
   // row order (x-direction) conductivity array 
  arg->sigmap = (float *)malloc(N * sizeof(float));

  // column order (y-direction) conductivity array 
  arg->sigmapy = (float *)malloc(N * sizeof(float));

  assert(arg->hostPP != NULL  && arg->sigmap != NULL && arg->sigmapy != NULL);
    
  memset(arg->sigmap,  0, N * sizeof(float));
  memset(arg->sigmapy, 0, N * sizeof(float));
  
  int idx2;
  for(k=0; k<arg->Nz1; k++) {
    for(j=0; j<arg->Ny1; j++) {
      for(i=0; i<arg->Nx1; i++) {
	idx0 = k*arg->Nx1*arg->Ny1 + j*arg->Nx1 + i;
	idx1 = k*Nx*Ny + j*Nx + i;
	idx2 = k*Nx*Ny + i*Ny + j;
    
	arg->sigmap[idx1]  = *arg->grid[idx0].sigmap;
	arg->sigmapy[idx2] = *arg->grid[idx0].sigmap;
      }
    }
  }
  
  arg->Nx = Nx;
  arg->Ny = Ny;
  arg->Nz = Nz;
  arg->N = N;

  // set parameters
  float tau = (arg->Hx+arg->Hy+arg->Hz)/12.0;
  float varying_tau = arg->time_step*(1.0/tau);
  arg->varying_tau = varying_tau;

  float H1 = 0.5/(arg->Hx*arg->Hx);
  float H2 = 0.5/(arg->Hy*arg->Hy);
  float H3 = 0.5/(arg->Hz*arg->Hz);

  arg->H1 = H1;
  arg->H2 = H2;
  arg->H3 = H3;

  // allocate device memory     
  assert(hipMalloc( (void**) &arg->PP,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Px,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Py,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Pz,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Rx,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Ry,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->Rz,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->sig,     N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->sigy,    N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->F0,      N*sizeof(float)) == hipSuccess);
  assert(hipMalloc( (void**) &arg->PP_diff, N*sizeof(float)) == hipSuccess);
  checkCUDAErrorBmd("memory allocation");
    
  hipMemcpy(arg->sig, arg->sigmap, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(arg->sigy, arg->sigmapy, N*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAErrorBmd("initialization");
  return 0; 
}

void free_memory_bmd(SolverParametersBmd *sp){

  hipFree(sp->PP);
  hipFree(sp->Px);
  hipFree(sp->Py);
  hipFree(sp->Pz);
  hipFree(sp->Rx);
  hipFree(sp->Ry);
  hipFree(sp->Rz);
  hipFree(sp->sig);
  hipFree(sp->sigy);
  hipFree(sp->F0);
  hipFree(sp->PP_diff);

  free(sp->hostPP);
  free(sp->sigmap);
  free(sp->sigmapy);
  
}
  
int multipl16_array_bmd(SolverParametersBmd *sp, int pos){
  
  int z = pos/(sp->Nx1*sp->Ny1);
  int rem = pos%(sp->Nx1*sp->Ny1);
  int y = rem/sp->Nx1;
  int x = rem%sp->Nx1;

  return (z*sp->Nx*sp->Ny + y*sp->Nx + x);
}

void update_conds(SolverParametersBmd *arg ) {

  for(int k=0; k<arg->Nz1; k++) {
    for(int j=0; j<arg->Ny1; j++) {
      for(int i=0; i<arg->Nx1; i++) {
	int idx0 = k*arg->Nx1*arg->Ny1 + j*arg->Nx1 + i;
	int idx1 = k*arg->Nx*arg->Ny + j*arg->Nx + i;
	int idx2 = k*arg->Nx*arg->Ny + i*arg->Ny + j;
    
	arg->sigmap[idx1]  = *arg->grid[idx0].sigmap;
	arg->sigmapy[idx2] = *arg->grid[idx0].sigmap;
      }
    }
  }
  hipMemcpy(arg->sig,  arg->sigmap,  arg->N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(arg->sigy, arg->sigmapy, arg->N*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAErrorBmd("UpdateCond");
}

extern "C" 
int solve_poisson_equation_cuda_bmd(GridPoint *grid, int Nx1, int Ny1, int Nz1, 
				float Hx, float Hy, float Hz, 
				float time_step, float tol, int Kmax, int device, 
				float *tissueConds, int num_tissues, int *srcPos, 
				float *srcVal, bool updateCond, int done)
{

  static SolverParametersBmd *sp = NULL;

  if (done){
    if (sp != NULL){
      free_memory_bmd(sp);
      free(sp);
      sp = NULL;
    }
    return 0;
  }


  if (sp == NULL){
    sp = (SolverParametersBmd*) malloc(sizeof(SolverParametersBmd));
    sp->grid = grid;
    sp->Nx1 = Nx1;
    sp->Ny1 = Ny1;
    sp->Nz1 = Nz1;
    sp->Hx  = Hx;
    sp->Hy  = Hy;
    sp->Hz  = Hz;
    sp->time_step = time_step;
    sp->tol = tol;
    sp->Kmax = Kmax;
    sp->device = device;
    sp->tissueConds = tissueConds;
    InitializeSolverBmd(sp);
    updateCond = false;
  }

  if (updateCond){
    printf("Updating cond "); fflush(stdout);
    update_conds(sp);
  }

  if (num_tissues > MAX_CONDS) {
    fprintf(stderr, "Error copying tissues conds to GPU ... number of tissues exceeds MAX_CONDS \n");
    return 0;
  }

  // set block sizes
  dim3 dimBlock_F0x(16, 16);
  dim3 dimGrid_F0x(sp->Nx/dimBlock_F0x.x, sp->Ny*sp->Nz/dimBlock_F0x.y);

  dim3 dimBlock_x1(16, 16);
  dim3 dimGrid_x1(sp->Ny/dimBlock_x1.x, sp->Nz/dimBlock_x1.y);
    
  dim3 dimBlock_x2(sp->Nx1);
  dim3 dimGrid_x2(sp->Ny1, sp->Nz1);
    
  dim3 dimBlock_y1(16, 16);
  dim3 dimGrid_y1(sp->Nx/dimBlock_y1.x, sp->Nz/dimBlock_y1.y);
    
  dim3 dimBlock_y2(sp->Nx1);
  dim3 dimGrid_y2(Ny1-2, Nz1-2);

  dim3 dimBlock_z1(16, 16);
  dim3 dimGrid_z1(sp->Nx/dimBlock_z1.x, sp->Ny/dimBlock_z1.y);
    
  dim3 dimBlock_z2(sp->Nx1);
  dim3 dimGrid_z2(sp->Ny1, sp->Nz1-2);

 // copy data to device memory 
  hipMemset(sp->PP, 0, sp->N*sizeof(float));
  hipMemset(sp->Px, 0, sp->N*sizeof(float));
  hipMemset(sp->Py, 0, sp->N*sizeof(float));
  hipMemset(sp->Pz, 0, sp->N*sizeof(float));
  hipMemset(sp->Rx, 0, sp->N*sizeof(float));
  hipMemset(sp->Ry, 0, sp->N*sizeof(float));
  hipMemset(sp->Rz, 0, sp->N*sizeof(float));
  hipMemset(sp->PP_diff, 0, sp->N*sizeof(float));
  checkCUDAErrorBmd("initialization");
        
  int   iter = 1;
  float pp_diff = FLT_MAX;

  int   src1  = multipl16_array_bmd(sp, srcPos[0]);
  int   src2  = multipl16_array_bmd(sp, srcPos[1]);
  float srcv1 = srcVal[0];
  float srcv2 = srcVal[1];

  do {

    SetF0XBmd <<< dimGrid_F0x, dimBlock_F0x >>> (sp->F0, sp->PP, sp->Ry, sp->Rz, sp->varying_tau, 
					      sp->Nx, sp->Ny*sp->Nz, src1, src2, srcv1, srcv2);
    checkCUDAErrorBmd("SetF0XBmd");

    SolveX1Bmd <<< dimGrid_x1, dimBlock_x1 >>> (sp->F0, sp->Px, sp->sigy, sp->Nx, sp->Ny, sp->Nz, 
					     sp->Nx1, sp->Ny1, sp->Nz1, sp->H1, sp->varying_tau);
    checkCUDAErrorBmd("SolveX1Bmd");

    SolveX2Bmd <<< dimGrid_x2, dimBlock_x2 >>> (sp->Px, sp->Rx, sp->sig, sp->Nx, sp->Ny, sp->Nz, 
					     sp->Nx1, sp->Ny1, sp->Nz1, sp->H1);
    checkCUDAErrorBmd("SolveX2Bmd");

    SolveY1Bmd <<< dimGrid_y1, dimBlock_y1 >>> (sp->PP, sp->Py, sp->Rx, sp->Rz, sp->sig, sp->Nx, 
					     sp->Ny, sp->Nz, sp->Nx1, sp->Ny1, sp->Nz1, sp->H2, 
    					     sp->varying_tau, src1, src2, srcv1, srcv2);
    checkCUDAErrorBmd("SolveY1Bmd");

    SolveY2Bmd <<< dimGrid_y2, dimBlock_y2 >>>  (sp->Ry, sp->Py, sp->sig, sp->Nx, sp->Ny, sp->Nz, 
					      sp->Nx1, sp->Ny1, sp->Nz1, sp->H2);
    checkCUDAErrorBmd("SolveY2Bmd");

    SolveZ1Bmd <<< dimGrid_z1, dimBlock_z1 >>> (sp->PP, sp->Px, sp->Py, sp->Pz, sp->Rx, sp->Ry, 
					     sp->sig, sp->Nx, sp->Ny, sp->Nz, sp->Nx1, sp->Ny1, 
					     sp->Nz1, sp->H3, sp->varying_tau, sp->PP_diff, src1, 
					     src2, srcv1, srcv2);
    checkCUDAErrorBmd("SolveZ1Bmd");

    SolveZ2Bmd <<< dimGrid_z2, dimBlock_z2 >>> (sp->Rz, sp->Pz, sp->sig, sp->Nx, sp->Ny, sp->Nz, sp->Nx1, 
					     sp->Ny1, sp->Nz1, sp->H3);
    checkCUDAErrorBmd("SolveZ2Bmd");
        
    pp_diff = full_reduce(sp->PP_diff, sp->PP_diff, sp->N, 512, 256);
    checkCUDAErrorBmd("reduce");

    hipMemset(sp->PP_diff, 0, sp->N*sizeof(float));


  } while(++iter <= sp->Kmax && pp_diff > sp->tol);

  //  printf("termination: %d %d %f %f \n", iter, sp->Kmax, pp_diff, sp->tol);
    
  // copy data back to host and free device memory
  // assert(hipMemcpy(sp->hostPP, sp->PP, sp->N*sizeof(float), hipMemcpyDeviceToHost)  == hipSuccess);

  hipMemcpy(sp->hostPP, sp->PP, sp->N*sizeof(float), hipMemcpyDeviceToHost);
  checkCUDAErrorBmd("copy");

  if (iter < 3) 
    fprintf(stderr, "Error computing on device %d \n", device);


  int k, j, i, idx0, idx1;

  // copy solution from flat array back into array of structs
  for(k=0; k<sp->Nz1; k++) {
    for(j=0; j<sp->Ny1; j++) {
      for(i=0; i<sp->Nx1; i++) {
	idx0 = k*sp->Nx1*sp->Ny1 + j*sp->Nx1 + i;
	idx1 = k*sp->Nx*sp->Ny + j*sp->Nx + i;
	sp->grid[idx0].PP = sp->hostPP[idx1];
	if (isnan(sp->grid[idx0].PP)){
	  fprintf(stderr, "Solve_adi_cuda: Invalid solution\n");
	  exit(1);
	}
      }
    }
  }

  //   printf("Iterative Loop time = %d %f\n ", iter, getWallTime()-t0);

  checkCUDAErrorBmd("finalization");
  return iter;
}
